#include "hip/hip_runtime.h"
//scan.cu
#include "comm.h"
#include "cuUtil.cu"
#include "graph.h"
#include "wtime.h"
#include "gputimer.h" //
#include "iostream"
#define max_thd 256 
#define max_block 256 

using namespace std;


__global__ void warp_binary_kernel
(	
	vertex_t*	adj_list,
	index_t*	begin,
	EDGE*		edge,
	index_t		Ns,
	index_t		Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = edge[tid].A;
		vertex_t B = edge[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}



__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void graph::scan(){


	vertex_t*	dev_adj;
	index_t*	dev_begin;
	EDGE*		dev_edge;
	index_t*	dev_count;
	GpuTimer gpu_timer; //

	H_ERR(hipMalloc(&dev_adj, edge_count*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_edge,  (edge_count)*sizeof(EDGE)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

		
	H_ERR(hipMemcpy(dev_adj,    adj_list, 	edge_count*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  beg_pos,  	(vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_edge,   edge, 	edge_count*sizeof(EDGE), hipMemcpyHostToDevice) );



	double time1=wtime(); //?
	//gpu_timer.Start(); //
	H_ERR(hipDeviceSynchronize() );
	warp_binary_kernel<<<max_block,max_thd>>>
	(	
		dev_adj,
		dev_begin,
		dev_edge,
		0,
		edge_count,
		dev_count
	);
	H_ERR(hipDeviceSynchronize() );
	reduce_kernel2 <<<1,1>>>(dev_count);
	H_ERR(hipDeviceSynchronize() );
	
	H_ERR(hipMemcpy(count, dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		

	double time2 = wtime();
	//gpu_timer.Stop(); //
	cout<<"GPU processing wtime = "<<time2-time1<<endl;
	cout<<"GPU processing gputime = "<<gpu_timer.Elapsed()<<endl;//
	cout<<"GPU triangle count = "<<count[0]<<endl;


	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_edge) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(dev_count) );
	return;	
}



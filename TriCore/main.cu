#include "graph.h"
#include "wtime.h"

#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
//	pthread_t *thd = new pthread_t[GPU_NUM];
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 		= new graph	(json_file); 
double t0=wtime();
//	graph_d->validation();
for (int i = 0; i<1; i++)	
	graph_d->scan();
double t1=wtime();
	cout<<"total time = "<<t1-t0<<" secondes"<<endl;

	return 0;
}
